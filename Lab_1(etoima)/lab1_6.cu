/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <sys/time.h>


#define FILTER_RADIUS 	8
#define FILTER_LENGTH 	(2 * FILTER_RADIUS + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	5e-14
////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }   
        h_Dst[y * imageW + x] = sum;  
      }
    }
  }
   
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH, int filterR) {

  
  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        } 
        h_Dst[y * imageW + x] = sum;  
      }
    }
  }

 
    
}

///// Reference row convolution filter in GPU   /////

__global__ void row_Kernel(double *Dst, double *Src, double *Filter,int imageW, int imageH, int filterR) {

     int k;
 
     double sum=0; //value to store the element of the matrix that is computed by the thread

        for( k= -filterR; k <= filterR; k++){
         int d = blockIdx.x * blockDim.x + threadIdx.x + k;

          if(d >= 0 && d < imageW) {
            sum += Src[(blockIdx.y * blockDim.y + threadIdx.y) * imageW + d] * Filter[filterR - k];    
          }	    
          Dst[(blockIdx.y * blockDim.y + threadIdx.y) * imageW + blockIdx.x * blockDim.x + threadIdx.x]=sum;
         }
     
       
   
   
}

///// Reference column convolution filter in GPU  /////


__global__ void column_Kernel(double *Dst, double *Src, double *Filter,int imageW, int imageH, int filterR) {

     int k;

  
   
      double sum=0; //value to store the element of the matrix that is computed by the thread

      for( k= -filterR; k <= filterR; k++){
	int d =blockIdx.y * blockDim.y + threadIdx.y+ k;

        if(d >= 0 && d < imageH) {
           sum += Src[d * imageW + blockIdx.x * blockDim.x + threadIdx.x] * Filter[filterR - k];    
        }	
        Dst[(blockIdx.y * blockDim.y + threadIdx.y) * imageW + blockIdx.x * blockDim.x + threadIdx.x]=sum;  
       }
  
        

}




////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *d_Input,     //eikona eisodou sto device
    *d_OutputGPU,  //apotelesma apo to device gpu
    *d_Filter,    //filtro sto device 
    *h_OutputGPU,      //to epistrefomeno apotelesma apo thn gpu sto host
    *d_Buffer;       //Buffer sto device gia endiameso apotelesma apo thn row sth column ston kernel

  

    int imageW;
    int imageH;
    unsigned int i;
    hipEvent_t start_GPU;  //var gia na metrisw xrono sth gpu
    hipEvent_t stop_GPU;   //var gia na metrisw xrono sth gpu 
    float elapsed_GPU;  //xronos sth gpu
    timeval t1;     //gia na metrisw to xrono sth cpu
    timeval t2;     //gia na metrisw to xrono sth cpu
    double elapsed_CPU;  // xronos sth cpu


    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputGPU = (double *)malloc(imageW * imageH * sizeof(double));

    

    // Allocate memory on device
    hipMalloc((void**)&d_Input, imageW * imageH * sizeof(double));
    hipMalloc((void**)&d_OutputGPU, imageW * imageH * sizeof(double));
    hipMalloc((void**)&d_Filter, ((2*FILTER_RADIUS)+1)*sizeof(double)); 
    hipMalloc((void**)&d_Buffer, imageW * imageH * sizeof(double));


    // if either memory allocation failed, report an error message
    if(h_Filter == 0 || h_Input == 0 || h_Buffer == 0 || h_OutputCPU == 0 || h_OutputGPU == 0 || d_Input ==0 
    || d_OutputGPU == 0 || d_Filter == 0 || d_Buffer == 0)
    {
     printf("couldn't allocate memory\n");
     return 1;
    }
    

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

   //arxikopoiw me floats wste na exw megaluterh anakriveia sta apotelesmata mou
   srand(time(NULL));
       	
    for (i = 0; i < FILTER_LENGTH; i++)
    {
        h_Filter[i] = (double)(rand() / (double)RAND_MAX);
	
		
    }

    
    for (i = 0; i < imageW * imageH; i++)
    {
        h_Input[i] = (double)(rand() / (double)RAND_MAX);
	
    }

      
     
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    ///// cpu events gia metrisi xronou  /////
     
     gettimeofday(&t1, NULL);

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, FILTER_RADIUS); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, FILTER_RADIUS); // convolution kata sthles

    ///// cpu events gia metrisi xronou  /////
    gettimeofday(&t2, NULL);
    elapsed_CPU = (t2.tv_sec - t1.tv_sec) +  ((t2.tv_usec - t1.tv_usec)/1000000.0);
    printf("CPU elapsed time:%f sec\n",elapsed_CPU);

    

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  

    
    printf("GPU computation...\n");


    /////  cuda events for gpu time calculation /////
    hipEventCreate(&start_GPU);
    hipEventCreate(&stop_GPU);
    hipEventRecord(start_GPU, 0);
    


  

    //Load h_Input and h_Filter to device memory
     hipMemcpy(d_Input,h_Input,imageW * imageH * sizeof(double),hipMemcpyHostToDevice);
     hipMemcpy(d_Filter,h_Filter,((2*FILTER_RADIUS)+1)*sizeof(double),hipMemcpyHostToDevice);
     
    // Kernel Invocation
    // Setup the execution configuration
    dim3 dimGrid;      
    dim3 dimBlock;
    dimBlock.x=4;
    dimBlock.y=4;
    dimGrid.x=imageW/dimBlock.x;
    dimGrid.y=imageH/dimBlock.y;
    
    

    //Launch the device
    hipDeviceSynchronize();
    row_Kernel<<<dimGrid,dimBlock>>>(d_Buffer,d_Input,d_Filter,imageW,imageH,FILTER_RADIUS);
    hipDeviceSynchronize();
    column_Kernel<<<dimGrid,dimBlock>>>(d_OutputGPU,d_Buffer,d_Filter,imageW,imageH,FILTER_RADIUS);
    hipDeviceSynchronize();


    // ask CUDA for the last error to occur (if one exists)
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
   
    printf("CUDA Error: %s\n", hipGetErrorString(error));

    
    return 1;
    } 

    //Read d_OutputGPU from the device
    hipMemcpy(h_OutputGPU,d_OutputGPU,imageW * imageH * sizeof(double),hipMemcpyDeviceToHost);


    /////  cuda events for gpu time calculation  /////
    hipEventRecord(stop_GPU, 0);
    hipEventSynchronize(stop_GPU);
    hipEventElapsedTime(&elapsed_GPU, start_GPU, stop_GPU);

    hipEventDestroy(start_GPU);
    hipEventDestroy(stop_GPU);


  

    printf("GPU elapsed time:%f sec\n ",elapsed_GPU/1000);

    

    printf("1.GPU:%d=%f\n",imageW * imageH-1,h_OutputGPU[imageW * imageH-1]); 
    printf("2.CPU:%d=%f\n",imageW * imageH-1,h_OutputCPU[imageW * imageH-1]);  
      

    
    
    // CPU Vs GPU (comparison) //
    

    for(i = 0; i< imageW * imageH; i++){
       if ( ABS(h_OutputCPU[i] - h_OutputGPU[i]) >= accuracy ) {
          printf("ERROR at element i:%d , accuracy error so i have to terminate sorry \n",i);
          return 1;
       }

    }
     
    
    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_OutputGPU);
    hipFree(d_Input);
    hipFree(d_Filter);
    hipFree(d_OutputGPU);
    hipFree(d_Buffer);



     printf("success !!!! \n");

    


    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
     hipDeviceReset();


    return 0;
}
